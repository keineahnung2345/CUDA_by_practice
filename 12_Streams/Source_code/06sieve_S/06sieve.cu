#include "hip/hip_runtime.h"
#include "common/Error.h"
#include "common/GpuTimer.h"
#include "common/Vector.h"

#define MARK 1
#define UNMARK 0
#define ARRAY_SIZE 16384

const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);


__global__ void kernelSieve(int k, Vector<int> d_a)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	while(i < ARRAY_SIZE)
	{
		if(k*k <= i)
                //the check starts from k*k, 
                //because k*1 is the multiple of 1, k*2 is the multiple of 2, ..., and k*(k-1) is the multiple of k-1,
                //they are all checked in previous iterations
		{
			if(i%k == 0)
                        //k's multiples are marked
				d_a.setElement(i, MARK);
		}
		i+=blockDim.x*gridDim.x;
	}
}


void onDevice(Vector<int> h_a)
{
	Vector<int> d_a;
	int k;
	// create the stream
	hipStream_t stream1;	
	HANDLER_ERROR_ERR(hipStreamCreate (&stream1));	

	HANDLER_ERROR_ERR(hipMalloc(&d_a.elements, ARRAY_BYTES));
	HANDLER_ERROR_ERR(hipMemcpy(d_a.elements, h_a.elements, ARRAY_BYTES, hipMemcpyHostToDevice));
	
	for( k=2; k<=ARRAY_SIZE; k++)
	{
		kernelSieve<<<64, 256, 0, stream1 >>>(k, d_a);
	}

	HANDLER_ERROR_ERR(hipMemcpy(h_a.elements, d_a.elements, ARRAY_BYTES, hipMemcpyDeviceToHost));
	HANDLER_ERROR_ERR(hipFree(d_a.elements));

	//destroy stream
	HANDLER_ERROR_ERR(hipStreamDestroy(stream1));		
}

void onHost(){

    Vector<int> h_a;
    h_a.length = ARRAY_SIZE;

	int j;
	h_a.elements = (int*)malloc( ARRAY_BYTES );


	for(j=0; j<ARRAY_SIZE; j++){
		h_a.setElement(j,j);
	}

	onDevice(h_a);
	
	for(j=0; j<ARRAY_SIZE; j++){
                //h_a.getElement(0) is 0 and h_a.getElement(1) is 1 so they are passed
                //composite numbers are also passed because they are marked as MARK=1
		if(h_a.getElement(j) > 1)
			printf("%i \n", h_a.getElement(j));
	}

	free(h_a.elements);
}

void checkDeviceProps(){
	//properties validation
    hipDeviceProp_t  prop;
    int whichDevice;
    HANDLER_ERROR_ERR( hipGetDevice( &whichDevice ) );
    HANDLER_ERROR_ERR( hipGetDeviceProperties( &prop, whichDevice ) );
    if (!prop.deviceOverlap) {
        printf( "Device will not handle overlaps, so no speed up from streams\n" );
  	}
 }


int main()
{
	checkDeviceProps();
	onHost();
}

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "common/Error.h"
#include "common/Matrix.h"
#include "common/GpuTimer.h"
#include "common/CpuTimer.h"

#define N 1000
#define K 16

void compareResults(Matrix<int> h_a, Matrix<int> h_b){

	int i,j;

	for(i=0; i < h_a.width; i++){
		for(j = 0; j < h_a.height; j++){
			assert(h_a.elements[ j * h_a.width + i ]  == h_b.elements[ i * h_b.width + j ]  );
		}
	}
}


__global__ void transposedMatrixKernel_pitch(Matrix<int> d_a, Matrix<int> d_b){

	int i = 0;
	int j = 0;

	while( i < d_a.width){
		j = 0;
		while( j < d_a.height){
			d_b.setElement( i, j, d_a.getElement(j, i) );
			j++;
		}
		i++;
	}
}


__global__ void transposedMatrixKernel(Matrix<int> d_a, Matrix<int> d_b){

	int i = 0;
	int j = 0;

	while( i < d_a.width){
		j = 0;
		while( j < d_a.height){
			d_b.setElement( i, j, d_a.getElement(j, i) );
			j++;
		}
		i++;
	}
	   
}


void onDevice(Matrix<int> h_a, Matrix<int> h_b){

    // declare GPU data
	Matrix<int> d_a, d_b;
	d_a.width = h_a.width;
	d_a.height = h_a.height;

	d_b.width = h_b.width;
	d_b.height = h_b.height;	


	GpuTimer timer;
	size_t pitchA;
	size_t pitchB;


	const int ARRAY_BYTES = d_a.width * d_a.height * sizeof(int);


    // -*- [Pitch Allocation] -*-
    // allocate  memory on the GPU

        //https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__MEMORY.html#group__CUDART__MEMORY_1g32bd7a39135594788a542ae72217775c
        /*
        __host__​hipError_t hipMallocPitch ( void** devPtr, size_t* pitch, size_t width, size_t height )
        Allocates pitched memory on the device.
        Allocates at least width (in bytes) * height bytes of linear memory on the device and returns in *devPtr a pointer to the allocated memory. The function may pad the allocation to ensure that corresponding pointers in any given row will continue to meet the alignment requirements for coalescing as the address is updated from row to row. The pitch returned in *pitch by hipMallocPitch() is the width in bytes of the allocation. The intended usage of pitch is as a separate parameter of the allocation, used to compute addresses within the 2D array. Given the row and column of an array element of type T, the address is computed as:

‎    T* pElement = (T*)((char*)BaseAddress + Row * pitch) + Column;
        */
        //d_a.width * sizeof(int): the width needed in bytes
        //pitchA: the allocated width in bytes
	HANDLER_ERROR_ERR(hipMallocPitch((void **)(&d_a.elements), &pitchA, d_a.width * sizeof(int), d_a.height));
	HANDLER_ERROR_ERR(hipMallocPitch((void **)(&d_b.elements), &pitchB, d_b.width * sizeof(int), d_b.height));
	printf("pitch = %li\n", pitchA/sizeof(int) );

	//copy from host memory to device
        //http://yzhu84.blogspot.com/2012/05/cudamallocpitch-and-cudamemcpy2d.html
        //here should use hipMemcpy2D() instead?
	HANDLER_ERROR_ERR(hipMemcpy(d_a.elements, h_a.elements, ARRAY_BYTES, hipMemcpyHostToDevice));
	HANDLER_ERROR_ERR(hipMemcpy(d_b.elements, h_b.elements, ARRAY_BYTES, hipMemcpyHostToDevice));

  	timer.Start();	

	transposedMatrixKernel_pitch<<<1,1>>>( d_a, d_b );
	HANDLER_ERROR_MSG("kernel panic!!!");
   	timer.Stop(); 
    printf( "Time Device pitch:  %f ms\n", timer.Elapsed() );
    // copy data back from the GPU to the CPU
	HANDLER_ERROR_ERR(hipMemcpy(h_b.elements, d_b.elements, ARRAY_BYTES, hipMemcpyDeviceToHost));	
	
	compareResults(h_a, h_b);

    // free GPU memory
	HANDLER_ERROR_ERR(hipFree(d_a.elements));
	HANDLER_ERROR_ERR(hipFree(d_b.elements));


    // -*- [CUDA Malloc Allocation] -*-

    // allocate  memory on the GPU
	HANDLER_ERROR_ERR(hipMalloc((void**)&d_a.elements,ARRAY_BYTES));
	HANDLER_ERROR_ERR(hipMalloc((void**)&d_b.elements,ARRAY_BYTES));

    // copy data from CPU the GPU
	HANDLER_ERROR_ERR(hipMemcpy(d_a.elements, h_a.elements, ARRAY_BYTES, hipMemcpyHostToDevice));
	HANDLER_ERROR_ERR(hipMemcpy(d_b.elements, h_b.elements, ARRAY_BYTES, hipMemcpyHostToDevice));

  	timer.Start();
	transposedMatrixKernel<<<1,1>>>( d_a, d_b );
    HANDLER_ERROR_MSG("kernel panic!!!");
   	timer.Stop(); 
    printf( "Time Device threads and blocks:  %f ms\n", timer.Elapsed() );
    // copy data back from the GPU to the CPU
	HANDLER_ERROR_ERR(hipMemcpy(h_b.elements, d_b.elements, ARRAY_BYTES, hipMemcpyDeviceToHost));	
	compareResults(h_a, h_b);

    // free GPU memory
	HANDLER_ERROR_ERR(hipFree(d_a.elements));
	HANDLER_ERROR_ERR(hipFree(d_b.elements));

}


void onHost(){


	Matrix<int> h_a, h_b;
	h_a.width = N;
	h_a.height = N;

	h_b.width = N;
	h_b.height = N;

	h_a.elements = (int*)malloc(h_a.width  * h_b.height  * sizeof(int));
	h_b.elements = (int*)malloc(h_b.width  * h_b.height  * sizeof(int));


	int i,j,k=0;

	for( i = 0; i < h_a.width; i++){
		for( j = 0; j < h_a.height; j++){
			h_a.elements[j*h_a.width + i] = k;	
			h_b.elements[j*h_b.width + i] = 0;	
			k++;			
		}
	}

    // call device configuration
	onDevice(h_a,h_b);


    printf("-: successful execution :-\n");

    free(h_a.elements);
    free(h_b.elements);

}


int main(){

	onHost();
}

/*
pitch = 1024
Time Device pitch:  128.752060 ms
Time Device threads and blocks:  128.749695 ms
-: successful execution :-
*/
//Ref to:
//https://devtalk.nvidia.com/default/topic/1010074/cuda-programming-and-performance/bad-performance-using-mallocpitch-and-memcpy2d/post/5153032/#5153032
//:pitched allocation is useful on early GPU, but are of less significance on modern GPUs
